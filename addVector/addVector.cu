#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addVector(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    int numElements = 50000;
    size_t size = numElements * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    float *d_B = NULL;
    hipMalloc((void **)&d_B, size);

    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    addVector<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipGetLastError();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    err = hipFree(d_A);
    err = hipFree(d_B);
    err = hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

