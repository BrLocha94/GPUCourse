#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void atomicHistogram(int * Histogram, const int * data)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    int value = data[id];
    int Histovalue = DetectRange(value);
    atomicAdd(&Histogram[Histovalue], 1);
}

int main(void)
{
    int numElements = 1000; 
    int threadsPerBlock = 512; 
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    size_t size = numElements * sizeof(float);

    float *host_histogram = (float *)malloc(size);
    float *host_data = (float *)malloc(size);

    for (int i = 0; i < numElements; ++i)
    {
        host_histogram[i] = rand()/(float)RAND_MAX;
        host_data[i] = rand()/(float)RAND_MAX;
    }

    float *device_histogram = NULL;  hipMalloc((void **)&device_histogram, size);
    float *device_data = NULL;  hipMalloc((void **)&device_data, size);

    hipMemcpy(device_histogram, host_histogram, size, hipMemcpyHostToDevice);
    hipMemcpy(device_data, host_data, size, hipMemcpyHostToDevice);

    hipEvent_t start,stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    atomicHistogram<<<blocksPerGrid, threadsPerBlock>>>(device_histogram, device_data);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(host_histogram, device_histogram, size, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    double memXFers=5*4*COLUMNS*ROWS;
    memXFers/=1024*1024*1024;

    printf("GPU: %f ms bandwidth %g GB/s",ms, memXFers/(ms/1000.0));
    printf("\n CPU : %g ms bandwidth %g GB/s",mtime, memXFers/(mtime/1000.0));

    hipFree(device_histogram);
    hipFree(device_data);

    free(host_histogram);
    free(host_data);

    return 0;
}