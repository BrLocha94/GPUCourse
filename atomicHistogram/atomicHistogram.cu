#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void atomicHistogram(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    int numElements = 1000; 
    int threadsPerBlock = 512; 
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    size_t size = numElements * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    float *d_A = NULL;  hipMalloc((void **)&d_A, size);
    float *d_B = NULL;  hipMalloc((void **)&d_B, size);
    float *d_C = NULL;  hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipEvent_t start,stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    addVector<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    double memXFers=5*4*COLUMNS*ROWS;
    memXFers/=1024*1024*1024;

    printf("GPU: %f ms bandwidth %g GB/s",ms, memXFers/(ms/1000.0));
    printf("\n CPU : %g ms bandwidth %g GB/s",mtime, memXFers/(mtime/1000.0));

    err = hipFree(d_A);
    err = hipFree(d_B);
    err = hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}